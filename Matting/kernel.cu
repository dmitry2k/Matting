#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>

using namespace cv;
using namespace std;

#define GRID_SIZE 1024
#define BLOCK_SIZE 1024
#define LOW_PASS 30
#define HIGH_PASS 245

#define CHECK(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
			<< " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
		exit(1);															\
	} }

//�������� ��������� ���� �� �������
__global__ void greensceen(uchar *a, int n1, int m1, uchar *b, int n2, int m2)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int threadsNum = blockDim.x*gridDim.x;
	for (int i = id; i < n1*m1; i += threadsNum)
	{
		int cur_row = i / m1;
		int cur_col = i % m1;
		int new_i3 = 3 * (m2*cur_row + cur_col);
		int i3 = 3 * i;


		if (!(a[i3 + 1] > a[i3] && a[i3 + 1] > a[i3 + 2] && a[i3 + 1] > LOW_PASS && a[i3 + 1] < HIGH_PASS))
		{
			b[new_i3] = a[i3];
			b[new_i3 + 1] = a[i3 + 1];
			b[new_i3 + 2] = a[i3 + 2];
		}

	}
}

int main(void)
{
	VideoCapture cap("video1.avi"); // ����� � �������� �� ������� ����
	if (!cap.isOpened())
	{
		cout << "Error can't find the file" << endl;
	}
	VideoCapture cap2("video2.avi"); // �����-���
	if (!cap2.isOpened())
	{
		cout << "Error can't find the file2" << endl;
	}

	Mat frame, frame2;
	namedWindow("1", WINDOW_AUTOSIZE); // ���� � �������� �� ������� ����
	namedWindow("2", WINDOW_AUTOSIZE); // ���� � �����
	namedWindow("3", WINDOW_AUTOSIZE); // ���������

	/*
	VideoWriter outputVideo;
	outputVideo.open("video3.avi", cap2.get(CV_CAP_PROP_FOURCC), cap2.get(CV_CAP_PROP_FPS), Size(cap2.get(CV_CAP_PROP_FRAME_WIDTH), cap2.get(CV_CAP_PROP_FRAME_HEIGHT)), true);
	if (!outputVideo.isOpened())
	{
		cout << "Could not open the output video file\n";
		return -1;
	}
	*/

	int m1 = cap.get(CV_CAP_PROP_FRAME_WIDTH);
	int n1 = cap.get(CV_CAP_PROP_FRAME_HEIGHT);
	int m2 = cap2.get(CV_CAP_PROP_FRAME_WIDTH);
	int n2 = cap2.get(CV_CAP_PROP_FRAME_HEIGHT);
	uchar *dev_a, *dev_b;

	hipEvent_t stopCUDA;
	hipEventCreate(&stopCUDA);

	CHECK(hipMalloc(&dev_a, 3 * n1*m1 * sizeof(uchar)));
	CHECK(hipMalloc(&dev_b, 3 * n2*m2 * sizeof(uchar)));

	while (true)
	{
		if (!cap.read(frame)) break;
		if (!cap2.read(frame2)) break;
		imshow("1", frame);
		imshow("2", frame2);
		CHECK(hipMemcpy(dev_a, frame.data, 3 * n1*m1 * sizeof(uchar), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(dev_b, frame2.data, 3 * n2*m2 * sizeof(uchar), hipMemcpyHostToDevice));

		greensceen<<<GRID_SIZE, BLOCK_SIZE >>>(dev_a, n1, m1, dev_b, n2, m2);

		hipEventRecord(stopCUDA, 0);
		hipEventSynchronize(stopCUDA);

		CHECK(hipMemcpy(frame2.data, dev_b, 3 * n2*m2 * sizeof(uchar), hipMemcpyDeviceToHost));
		imshow("3", frame2);
		//outputVideo.write(frame2);
		waitKey(33);
	}

	CHECK(hipFree(dev_a));
	CHECK(hipFree(dev_b));

	system("pause");

	return 0;
}
